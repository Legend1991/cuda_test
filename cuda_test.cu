#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

extern "C" {
#include "acpc_server_code/game.h"
}

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

const uint32_t LENGTH = MAX_SUITS * MAX_RANKS;

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void cudaTest(uint8_t *Res, int numElements) 
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  uint8_t p2[2] = {makeCard(10, 1), makeCard(8, 1)};

  const uint16_t P1_HAND_SET_SIZE = 338; // 169 * 2 (two cards per hand)
  uint8_t p1HandSet[P1_HAND_SET_SIZE] = {
      0, 4, 0, 8, 0, 12, 0, 16, 0, 20, 0, 24, 0, 28, 0, 32, 0, 36, 0, 40, 0, 44, 0, 48, 4, 8, 4, 12, 4, 16, 4, 20, 4, 24, 4, 28, 4, 32, 4, 36, 4, 40, 4, 44, 4, 48, 8, 12, 8, 16, 8, 20, 8, 24, 8, 28, 8, 32, 8, 36, 8, 40, 8, 44, 8, 48, 12, 16, 12, 20, 12, 24, 12, 28, 12, 32, 12, 36, 12, 40, 12, 44, 12, 48, 16, 20, 16, 24, 16, 28, 16, 32, 16, 36, 16, 40, 16, 44, 16, 48, 20, 24, 20, 28, 20, 32, 20, 36, 20, 40, 20, 44, 20, 48, 24, 28, 24, 32, 24, 36, 24, 40, 24, 44, 24, 48, 28, 32, 28, 36, 28, 40, 28, 44, 28, 48, 32, 36, 32, 40, 32, 44, 32, 48, 36, 40, 36, 44, 36, 48, 40, 44, 40, 48, 44, 48, // Suited
      0, 1, 4, 5, 8, 9, 12, 13, 16, 17, 20, 21, 24, 25, 28, 29, 32, 33, 36, 37, 40, 41, 44, 45, 48, 49,                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                           // Pairs
      0, 5, 0, 9, 0, 13, 0, 17, 0, 21, 0, 25, 0, 29, 0, 33, 0, 37, 0, 41, 0, 45, 0, 49, 4, 9, 4, 13, 4, 17, 4, 21, 4, 25, 4, 29, 4, 33, 4, 37, 4, 41, 4, 45, 4, 49, 8, 13, 8, 17, 8, 21, 8, 25, 8, 29, 8, 33, 8, 37, 8, 41, 8, 45, 8, 49, 12, 17, 12, 21, 12, 25, 12, 29, 12, 33, 12, 37, 12, 41, 12, 45, 12, 49, 16, 21, 16, 25, 16, 29, 16, 33, 16, 37, 16, 41, 16, 45, 16, 49, 20, 25, 20, 29, 20, 33, 20, 37, 20, 41, 20, 45, 20, 49, 24, 29, 24, 33, 24, 37, 24, 41, 24, 45, 24, 49, 28, 33, 28, 37, 28, 41, 28, 45, 28, 49, 32, 37, 32, 41, 32, 45, 32, 49, 36, 41, 36, 45, 36, 49, 40, 45, 40, 49, 44, 49  // Unsuited
  };

  const uint16_t p1Idx = i * 2;

  uint8_t p1[2] = {p1HandSet[p1Idx], p1HandSet[p1Idx + 1]};

  const uint32_t DECK_SIZE = 48;
  uint8_t deck[DECK_SIZE];

  for (uint8_t c = 0, i = 0; c < LENGTH; ++c)
  {
    if (c == p1[0] || c == p1[1] || c == p2[0] || c == p2[1])
      continue;

    deck[i] = c;
    ++i;
  }

  uint32_t count = numElements / (P1_HAND_SET_SIZE / 2) * i;

  for (uint8_t f1Idx = 0; f1Idx < DECK_SIZE; ++f1Idx)
  {
    for (uint8_t f2Idx = f1Idx + 1; f2Idx < DECK_SIZE; ++f2Idx)
    {
      for (uint8_t f3Idx = f2Idx + 1; f3Idx < DECK_SIZE; ++f3Idx)
      {
        for (uint8_t tIdx = f3Idx + 1; tIdx < DECK_SIZE; ++tIdx)
        {
          for (uint8_t rIdx = tIdx + 1; rIdx < DECK_SIZE; ++rIdx)
          {
            Res[count] = count % 3;
            ++count;
          }
        }
      }
    }
  }
}

/**
 * Host main routine
 */
int main(void) {
  // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;

  // Print the vector length to be used, and compute its size
  int numElements = 289379376;
  size_t size = numElements * sizeof(uint8_t);
  printf("[Vector addition of %d elements]\n", numElements);

  // Allocate the host input vector A
  uint8_t *h_Res = (uint8_t *)malloc(size);

  // Verify that allocations succeeded
  if (h_Res == NULL) {
    fprintf(stderr, "Failed to allocate host vectors!\n");
    exit(EXIT_FAILURE);
  }

  // Initialize the host input vectors
  for (int i = 0; i < numElements; ++i) {
    h_Res[i] = -1;
  }

  clock_t start = clock();

  // Allocate the device input vector A
  uint8_t *d_Res = NULL;
  err = hipMalloc((void **)&d_Res, size);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Copy the host input vectors A and B in host memory to the device input
  // vectors in
  // device memory
  printf("Copy input data from the host memory to the CUDA device\n");
  err = hipMemcpy(d_Res, h_Res, size, hipMemcpyHostToDevice);

  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector A from host to device (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Launch the Vector Add CUDA Kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
         threadsPerBlock);
  cudaTest<<<1, 169>>>(d_Res, numElements);
  err = hipGetLastError();

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch cudaTest kernel (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Copy the device result vector in device memory to the host result vector
  // in host memory.
  printf("Copy output data from the CUDA device to the host memory\n");
  err = hipMemcpy(h_Res, d_Res, size, hipMemcpyDeviceToHost);

  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector C from device to host (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  clock_t end = clock();
  double diff = (double)(end - start) / (double)(CLOCKS_PER_SEC);

  printf("Calc took: %0.10f\n", diff);

  // Verify that the result vector is correct
  for (int i = 0; i < numElements; ++i) {
    if (h_Res[i] == -1) {
      fprintf(stderr, "Result verification failed at element %d!\n", i);
      exit(EXIT_FAILURE);
    }
  }

  printf("Test PASSED\n");

  // Free device global memory
  err = hipFree(d_Res);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector A (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Free host memory
  free(h_Res);

  printf("Done\n");

  return 0;
}
